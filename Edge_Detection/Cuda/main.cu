#include <stdio.h>


#include "functions.c"


//false --> No vertical gradient and horizontal gradient are output
//true --> Vertical gradient and horizontal gradient are output
#define INTERMEDIATE_OUTPUT false
#define SOBEL_OP_SIZE 9
#define STRING_BUFFER_SIZE 1024

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )
#define get_time(time) (gettimeofday(&time, NULL))


#include "string.h"
#include "stdlib.h"
#include "math.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

#include "kernels.cu"


static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
      {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE);
    }
}



int main ( int argc, char** argv )
{
		//all the time_val declarations are put at the beginning of the file for better code readability
		struct timeval comp_start_rgb_to_gray, comp_end_rgb_to_gray;
		struct timeval comp_start_horiz_grad, comp_end_horiz_grad;
		struct timeval comp_start_vert_grad, comp_end_vert_grad;
		struct timeval start_countour_alloc, end_countour_alloc;
		struct timeval start_countour_copy, end_countour_copy;
		struct timeval start_free_countour, end_free_countour;
		struct timeval i_o_start_write_img, i_o_end_write_img;





		//########### 1. STEP - LOAD THE IMAGE, ITS HEIGHT, WIDTH AND CONVERT IT TO RGB FORMAT #########

		//########### Loading Image ######################

		//Specify the input image. Formats supported: png, jpg, GIF.
		const char * file_output_rgb = "imgs_out/image.rgb";
		const char *png_strings[4] = {"convert ", argv[1], " ", file_output_rgb};
		const char * str_PNG_to_RGB = array_strings_to_string(png_strings, 4, STRING_BUFFER_SIZE);

		//########### Convertion Image (to RGB) ###########

		//execute the conversion from PNG to RGB, as that format is required by the program
		int status_conversion = system(str_PNG_to_RGB);
		// check if the conversion is suucessful
		if(status_conversion != 0)
		{
			printf("ERROR! Conversion of input PNG image to RGB was not successful. Program aborting.\n");
			return -1;
		}

		//get the height and width of the input image
		int width = 0;
		int height = 0;

		get_image_size(argv[1], &width, &height);

		//Three dimensions because the input image is in RGB format
		int rgb_size = width * height * 3;

		//Used as a buffer for all pixels of the image
		byte * rgb_image;

		//Load up the input image in RGB format into one single flattened array (rgbImage)
		read_file(file_output_rgb, &rgb_image, rgb_size);


		//######################## 2. step - convert RGB image to gray-scale ########################


	  int gray_size = rgb_size / 3;
	  byte * r_vector, * g_vector, * b_vector;

	  //now take the RGB image vector and create three separate arrays for the R,G,B dimensions
	  get_dimension_from_RGB_vec(0, rgb_image,  &r_vector, gray_size);
	  get_dimension_from_RGB_vec(1, rgb_image,  &g_vector, gray_size);
	    get_dimension_from_RGB_vec(2, rgb_image,  &b_vector, gray_size);

    //allocate memory on the device for the r,g,b vectors
    byte * dev_r_vec, * dev_g_vec, * dev_b_vec;
    byte * dev_gray_image;


    // memory allocation for cuda conversion computation
    HANDLE_ERROR ( hipMalloc((void **)&dev_r_vec, gray_size*sizeof(byte)));
    HANDLE_ERROR ( hipMalloc((void **)&dev_g_vec, gray_size*sizeof(byte)));
    HANDLE_ERROR ( hipMalloc((void **)&dev_b_vec, gray_size*sizeof(byte)));
    //copy the content of the r,g,b vectors from the host to the device (cuda)
    HANDLE_ERROR (hipMemcpy (dev_r_vec , r_vector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMemcpy (dev_g_vec , g_vector , gray_size*sizeof(byte), hipMemcpyHostToDevice));
    HANDLE_ERROR (hipMemcpy (dev_b_vec , b_vector, gray_size*sizeof(byte), hipMemcpyHostToDevice));
    //allocate memory on the device for the output gray image
    HANDLE_ERROR ( hipMalloc((void **)&dev_gray_image, gray_size*sizeof(byte)));

	  // starting time (cuda) - RGB to Grayscale computation
		get_time(comp_start_rgb_to_gray);

	  	//actually run the kernel to convert input RGB file to gray-scale
	  	rgb_img_to_gray <<< width, height>>> (dev_r_vec, dev_g_vec, dev_b_vec, dev_gray_image, gray_size) ;
	  	hipDeviceSynchronize();
		
			byte * gray_image = (byte *) malloc(gray_size * sizeof(byte));

	  // starting time (cuda) - RGB to Grayscale computation
		get_time(comp_end_rgb_to_gray);

	  //Now take the device gray vector and bring it back to the host
	  HANDLE_ERROR (hipMemcpy(gray_image , dev_gray_image , gray_size*sizeof(byte) , hipMemcpyDeviceToHost));

		char str_width[100];
		sprintf(str_width, "%d", width);

		char str_height[100];
		sprintf(str_height, "%d", height);

	  hipFree (dev_r_vec);
	  hipFree (dev_g_vec);
		hipFree (dev_b_vec);

		//###################### 3. Step - Compute vertical and horizontal gradient ##########
 
		//######### Compute the HORIZONTAL GRADIENT #########

   	//host horizontal kernel
		int sobel_h[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
		int * dev_sobel_h;
   	byte * dev_sobel_h_res;

		//allocate memory for device horizontal kernel
		HANDLE_ERROR ( hipMalloc((void **)&dev_sobel_h , SOBEL_OP_SIZE*sizeof(int)));
		//copy the content of the host horizontal kernel to the device horizontal kernel
	  HANDLE_ERROR (hipMemcpy (dev_sobel_h , sobel_h , SOBEL_OP_SIZE*sizeof(int) , hipMemcpyHostToDevice));
	  //allocate memory for the resulting horizontal gradient on the device
		HANDLE_ERROR ( hipMalloc((void **)&dev_sobel_h_res , gray_size*sizeof(byte)));


	  // starting time (cuda) - horizontal calculation
		get_time(comp_start_horiz_grad);

			//perform horizontal gradient calculation for every pixel
			it_conv <<< width, height>>> (dev_gray_image, gray_size, width, dev_sobel_h, dev_sobel_h_res);
	  	hipDeviceSynchronize();
			//fixed segmentation fault when processing large images by using a malloc
			byte* sobel_h_res = (byte*) malloc(gray_size * sizeof(byte));
		
		// ending time (cuda) - horizontal calculation
		get_time(comp_end_horiz_grad);

		//copy the resulting horizontal array from device to host
	  HANDLE_ERROR (hipMemcpy(sobel_h_res , dev_sobel_h_res , gray_size*sizeof(byte) , hipMemcpyDeviceToHost));

	  //free-up the memory for the vectors allocated
	  hipFree(dev_sobel_h);


		//######### Compute the VERTICAL GRADIENT #########

	  int sobel_v[] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
		int * dev_sobel_v;
		byte * dev_sobel_v_res;


		//allocate memory for device vertical kernel
		HANDLE_ERROR (hipMalloc((void **)&dev_sobel_v , SOBEL_OP_SIZE*sizeof(int)));
		//copy the content of the host vertical kernel to the device vertical kernel
		HANDLE_ERROR (hipMemcpy (dev_sobel_v , sobel_v , SOBEL_OP_SIZE*sizeof(int) , hipMemcpyHostToDevice));
		//allocate memory for the resulting vertical gradient on the device
		HANDLE_ERROR (hipMalloc((void **)&dev_sobel_v_res , gray_size*sizeof(byte)));

	  // starting time (cuda) - vertical calculation
		get_time(comp_start_vert_grad);

			//perform vertical gradient calculation for every pixel
			it_conv <<<width, height>>> (dev_gray_image, gray_size, width, dev_sobel_v, dev_sobel_v_res);
	    hipDeviceSynchronize();
			//copy the resulting vertical array from device back to host
			//fixed segmentation fault issue with big images
			byte* sobel_v_res = (byte*) malloc(gray_size * sizeof(byte));

	  // ending time (cuda) - vertical calculation
		get_time(comp_end_vert_grad);


		HANDLE_ERROR (hipMemcpy(sobel_v_res , dev_sobel_v_res , gray_size*sizeof(byte) , hipMemcpyDeviceToHost));

		//free-up the memory for the vectors allocated
		hipFree(dev_sobel_v);


		//#############4. Step - Compute the countour by putting together the vertical and horizontal gradients####
		//allocate device memory for the final vector containing the countour

		byte * dev_countour_img;


		get_time(start_countour_alloc);
		HANDLE_ERROR ( hipMalloc((void **)&dev_countour_img , gray_size*sizeof(byte)));
		get_time(end_countour_alloc);

		struct timeval comp_start_countour_merge, comp_end_countour_merge;

		get_time(comp_start_countour_merge);
		contour <<< width, height>>> (dev_sobel_h_res, dev_sobel_v_res, gray_size, dev_countour_img);
	    hipDeviceSynchronize();

		//copy the resulting countour image from the device back to host
		byte * countour_img = (byte *) malloc(gray_size * sizeof(byte));

		get_time(comp_end_countour_merge);

		get_time(start_countour_copy);
		HANDLE_ERROR (hipMemcpy(countour_img, dev_countour_img, gray_size*sizeof(byte) , hipMemcpyDeviceToHost));
		get_time(end_countour_copy);

		get_time(start_free_countour);
		//free-up all the memory from the allocated vectors
	    hipFree(dev_sobel_h_res);
	    hipFree(dev_sobel_v_res);
	    hipFree(dev_countour_img);
	    get_time(end_free_countour);

	    //######Display the resulting countour image

		get_time(i_o_start_write_img);
	    output_gradient(true, countour_img, gray_size, str_width, str_height, STRING_BUFFER_SIZE, "imgs_out/sobel_countour.png");
		get_time(i_o_end_write_img);


		//#############5. Step - Display the elapsed time in the different parts of the code

		//##GPU memory movements (hipMalloc, cudaMemCpy, hipFree) ##
		//countour image operations
		double time_alloc_countour = compute_elapsed_time(start_countour_alloc, end_countour_alloc);
		double time_copy_countour = compute_elapsed_time(start_countour_copy, end_countour_copy);
		double time_free_countour = compute_elapsed_time(start_free_countour, end_free_countour);

		//##Actual GPU computation##
		double comp_time_rgb_to_gray = compute_elapsed_time(comp_start_rgb_to_gray, comp_end_rgb_to_gray);
		double comp_time_h_grad = compute_elapsed_time(comp_start_horiz_grad, comp_end_horiz_grad);
		double comp_time_v_grad = compute_elapsed_time(comp_start_vert_grad, comp_end_vert_grad);
		double comp_time_count_merge = compute_elapsed_time(comp_start_countour_merge, comp_end_countour_merge);


		//##Input/Output over the disk (image loading and final image writing)##
		double i_o_time_write_img = compute_elapsed_time(i_o_start_write_img, i_o_end_write_img);

		//let's deallocate the heap memory to avoid any memory leaks
		free(gray_image);
		free(sobel_h_res);
		free(sobel_v_res);
		free(countour_img);

	    return 0;

}
